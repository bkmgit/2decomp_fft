//=======================================================================
// This is part of the 2DECOMP&FFT library
// 
// 2DECOMP&FFT is a software framework for general-purpose 2D (pencil) 
// decomposition. It also implements a highly scalable distributed
// three-dimensional Fast Fourier Transform (FFT).
//
// Copyright (C) 2009-2021 Ning Li, the Numerical Algorithms Group (NAG)
//
//=======================================================================

// This contains CUDA code that compute multiple 1D FFTs on NVidia GPU

#include <stdio.h>
#include <stdlib.h>
#include "hipfft/hipfft.h"
#include "hip/hip_runtime.h"

extern "C" void fft_1m_r2c_(int *nx, int *m, hipfftDoubleReal *h_a, hipfftDoubleComplex *h_b)
{
  unsigned long size1 = sizeof(hipfftDoubleReal) * (*nx)*(*m);
  unsigned long size2 = sizeof(hipfftDoubleComplex) * (*nx/2+1)*(*m);
  hipfftDoubleReal *d_ic = NULL;
  hipfftDoubleComplex *d_oc = NULL; 
  hipfftHandle plan;
  hipMalloc((void **)&d_ic, size1);
  hipMalloc((void **)&d_oc, size2);
  hipMemcpy(d_ic, h_a, size1, hipMemcpyHostToDevice);
  int dims[1] = {*nx};
  hipfftPlanMany(&plan,1,dims,NULL,1,0,NULL,1,0,HIPFFT_D2Z,*m);
  hipfftExecD2Z(plan, d_ic, d_oc);
  hipMemcpy(h_b, d_oc, size2, hipMemcpyDeviceToHost);
  hipFree(d_ic);
  hipFree(d_oc);
  hipfftDestroy(plan);
}


extern "C" void fft_1m_c2r_(int *nx, int *m, hipfftDoubleComplex *h_a, hipfftDoubleReal *h_b)
{
  unsigned long size1 = sizeof(hipfftDoubleComplex) * (*nx/2+1)*(*m);
  unsigned long size2 = sizeof(hipfftDoubleReal) * (*nx)*(*m);
  hipfftDoubleComplex *d_ic = NULL;
  hipfftDoubleReal *d_oc = NULL; 
  hipfftHandle plan;
  hipMalloc((void **)&d_ic, size1);
  hipMalloc((void **)&d_oc, size2);
  hipMemcpy(d_ic, h_a, size1, hipMemcpyHostToDevice);
  int dims[1] = {*nx};
  hipfftPlanMany(&plan,1,dims,NULL,1,0,NULL,1,0,HIPFFT_Z2D,*m);
  hipfftExecZ2D(plan, d_ic, d_oc);
  hipMemcpy(h_b, d_oc, size2, hipMemcpyDeviceToHost);
  hipFree(d_ic);
  hipFree(d_oc);
  hipfftDestroy(plan);
}


extern "C" void fft_1m_c2c_(int *nx, int *m, hipfftDoubleComplex *h_a, hipfftDoubleComplex *h_b, int *sign)
{
  unsigned long size1 = sizeof(hipfftDoubleComplex) * (*nx)*(*m);
  hipfftDoubleComplex *d_ic = NULL;
  hipfftDoubleComplex *d_oc = NULL; 
  hipfftHandle plan;
  hipMalloc((void **)&d_ic, size1);
  hipMalloc((void **)&d_oc, size1);
  hipMemcpy(d_ic, h_a, size1, hipMemcpyHostToDevice);
  int dims[1] = {*nx};
  hipfftPlanMany(&plan,1,dims,NULL,1,0,NULL,1,0,HIPFFT_Z2Z,*m);
  hipfftExecZ2Z(plan, d_ic, d_oc, *sign);
  hipMemcpy(h_b, d_oc, size1, hipMemcpyDeviceToHost);
  hipFree(d_ic);
  hipFree(d_oc);
  hipfftDestroy(plan);
}
